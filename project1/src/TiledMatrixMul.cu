#include<hip/hip_runtime.h>

#include<iostream>
#include<fstream>
#include<string>
#include<sstream>
#include<memory>
#include<parseOprand.hpp>
#include<spdlog/spdlog.h>

#include<TiledMatrixMul_kernel.cu>


#ifndef USING_TILE
#define USING_TILE 1
#endif


using namespace std;


#ifndef blockSize
#define blockSize 32//real size is blockSize*blockSize
#endif

int main(int argc, char const *argv[])
{

    spdlog::set_pattern("[%c] [%@] [%^-%L-%$] %v");
    #if USING_TILE==1
    SPDLOG_INFO("USING THE TILE ALGOR");
    #else
    SPDLOG_INFO("USING NORMAL ALGOR");
    #endif
    
    SPDLOG_INFO("the blocksize is {}",blockSize);
    int row;
    
    
    if(0!=parseOpt(argc,argv,row)){
        SPDLOG_ERROR("parseOpt false");
        return -1;
    }
    SPDLOG_DEBUG("the row={} ",row);
    float* martrix_A;
    float* martrix_B;
    float* martrix_output;
    try
    {
        martrix_A=new float[row*row];
        martrix_B=new float[row*row];
        martrix_output=new float[row*row];
    }
    catch(const std::bad_alloc& e)
    {
        SPDLOG_ERROR( e.what() );
        return -1;
    }
    
    


    dim3 blockDim(blockSize,blockSize);
    int gridsize=(row+blockSize-1)/blockSize;
    dim3 gridDim(gridsize,gridsize);
    SPDLOG_DEBUG("grid size = {}",gridsize);


    //init martrix
    for(int i=0;i<row*row;i++){
        martrix_A[i]=1.0;
        martrix_B[i]=1.0;
    }

    float *d_a;
    float *d_b;
    float *d_out;
    if(hipSuccess!=hipMalloc(&d_a,row*row*sizeof(float))){
        SPDLOG_ERROR("cannot allocate device memory");
        return -1;
    }
    if(hipSuccess!=hipMalloc(&d_b,row*row*sizeof(float))){
        SPDLOG_ERROR("cannot allocate device memory");
        return -1;
    }
    if(hipSuccess!=hipMalloc(&d_out,row*row*sizeof(float))){
        SPDLOG_ERROR("cannot allocate device memory");
        return -1;
    }
    if(hipSuccess!=hipMemcpy(d_a,martrix_A,sizeof(float)*row*row,hipMemcpyHostToDevice)){
        SPDLOG_ERROR("cannot move memory");
        return -1;
    }
    if(hipSuccess!=hipMemcpy(d_b,martrix_B,sizeof(float)*row*row,hipMemcpyHostToDevice)){
        SPDLOG_ERROR("cannot move memory");
        return -1;
    }
    SPDLOG_INFO("start to lauch kernel ,wait");
    hipEvent_t start[3], stop[3];
    for(int i=0;i<3;i++){
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
    } 
    hipEventRecord(start[0]);
    #if USING_TILE==1
    tile<blockSize><<<gridDim,blockDim>>>(row,d_a,d_b,d_out);
    #else
    normal<<<gridDim,blockDim>>>(row,d_a,d_b,d_out);
    #endif
    hipEventRecord(stop[0]);

    
    
    if(hipGetLastError()!=hipSuccess){
        SPDLOG_ERROR("cannot launch kernel");
        return -1;
    }
    if(hipSuccess!=hipMemcpy(martrix_output,d_out,sizeof(float)*row*row,hipMemcpyDeviceToHost)){
        SPDLOG_ERROR("cannot retrive data");
        return -1;
    }

    SPDLOG_INFO("finish excute,start varify");
    for(int i=0;i<row*row;i++){
        if(martrix_output[i]!=row){
            SPDLOG_DEBUG("answer not correct");
            SPDLOG_DEBUG("output dumped");
            ofstream out("dump.txt");
            for(int j=0;j<row;j++){
                for(int k=0;k<row;k++){
                    out<<martrix_output[j*row+i]<<" ";
                }
                out<<endl;
            }
            return -1;
        }

    }

    SPDLOG_INFO("finished varify,all passed");
    hipEventSynchronize(stop[0]);
    float time=0;
    hipEventElapsedTime(&time,start[0],stop[0]);
    #if USING_TILE==1
    SPDLOG_INFO("the elapsed time of TILED with block size {} is {}",blockSize,time);
    #else
    SPDLOG_INFO("the elapsed time of NORMAL with block size {} is {}",blockSize,time);
    #endif


   
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    delete[] martrix_A;
    delete[] martrix_B;
    delete[] martrix_output;








    

    
    
}
