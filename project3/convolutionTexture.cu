#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef OLD
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#define BLKSIZE 32
#include "convolutionTexture_common.h"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
//Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

//Use unrolled innermost convolution loop
#define UNROLL_INNER 1

//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b)
{
    return (a % b != 0) ? (a - a % b + b) : a;
}



////////////////////////////////////////////////////////////////////////////////
// Convolution kernel and input array storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[128];

extern "C" void setConvolutionKernel(float *h_Kernel,int kernel_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, kernel_length * sizeof(float));
}

texture<float, 2, hipReadModeElementType> texSrc;

extern "C" void setInputArray(hipArray *a_Src)
{
}

extern "C" void detachInputArray(void)
{
}



////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template<int i> __device__ float convolutionRow(float x, float y)
{
    return
        tex2D(texSrc, x + (float)(KERNEL_RADIUS - i), y) * c_Kernel[i]
        + convolutionRow<i - 1>(x, y);
}

template<> __device__ float convolutionRow<-1>(float x, float y)
{
    return 0;
}

template<int i> __device__ float convolutionColumn(float x, float y)
{
    return
        tex2D(texSrc, x, y + (float)(KERNEL_RADIUS - i)) * c_Kernel[i]
        + convolutionColumn<i - 1>(x, y);
}

template<> __device__ float convolutionColumn<-1>(float x, float y)
{
    return 0;
}



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsKernel(
    #ifdef GB
    float *d_Input,
    #endif
    float *d_Dst,
    int imageW,
    int imageH,
     int kernel_size
)
{
    __shared__  float s[BLKSIZE];
    
    /*
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;
    */
    int radium=(kernel_size-1)>>1;
    int base_x=blockIdx.x*(blockDim.x-2*radium)-radium;//important!
    const int base_y=blockIdx.y*blockDim.y+threadIdx.y;
    #ifdef GB
    if(base_x +threadIdx.x<0)
    s[threadIdx.x]=d_Input[base_y*imageW];
    else if(base_x +threadIdx.x>=imageW)
    {
       s[threadIdx.x]=d_Input[base_y*imageW+imageW-1];
    }else
        s[threadIdx.x]=d_Input[base_y*imageW+base_x+threadIdx.x];
    #else
    s[threadIdx.x]=tex2D(texSrc, base_x+threadIdx.x+0.5 , base_y+0.5);
    #endif
    __syncthreads();
    int ix=base_x+threadIdx.x;
    const int iy=base_y;

    if (ix >= imageW )
    {
        return;
    }
    float sum=0;
    if(threadIdx.x>=radium && threadIdx.x <= blockDim.x-radium-1){//only middle thread attend compute
        for(int i=-radium;i<=radium;i++){
            sum += s[threadIdx.x+i] * c_Kernel[radium + i];//why it is -i from samples?
        }
    }
    

    

/*
#if(UNROLL_INNER)
    sum = convolutionRow<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x + (float)k, y) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif
*/

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}


extern "C" void convolutionRowsGPU(
    #ifdef GB
    float *d_Input,
    #endif
    float *d_Dst,
    #ifndef GB
    hipArray *a_Src,
    #endif
    int imageW,
    int imageH,
    int kernel_length
)
{
    dim3 threads(BLKSIZE);
    dim3 blocks(iDivUp(imageW, threads.x-kernel_length+1), iDivUp(imageH, threads.y));
    #ifndef GB
    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    #endif
    convolutionRowsKernel<<<blocks, threads>>>(
        #ifdef GB//that is global memory
        d_Input,
        #endif
        d_Dst,
        imageW,
        imageH,
        kernel_length
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
    #ifndef GB
    checkCudaErrors(hipUnbindTexture(texSrc));
    #endif

}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsKernel(
    #ifdef GB
    float *d_Input,
    #endif
    float *d_Dst,
    int imageW,
    int imageH,
    int kernel_length
)
{
    int radius=(kernel_length-1)>>1;
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)//divergency
    {
        return;
    }

    float sum = 0;



    for (int k = -radius; k <= radius; k++)
    {
        #ifdef GB
        int toY=iy+k;
        if(toY<0) toY=0;
        if(toY>=imageH) toY=imageH-1;
        sum+= d_Input[toY*imageW+ix]* c_Kernel[radius + k];
        #else
        sum += tex2D(texSrc, x, y + (float)k) * c_Kernel[radius + k];
        #endif
    }



    d_Dst[IMAD(iy, imageW, ix)] = sum;
}

extern "C" void convolutionColumnsGPU(
    #ifdef GB
    float *d_Input,
    #endif
    float *d_Dst,
    #ifndef GB
    hipArray *a_Src,
    #endif
    int imageW,
    int imageH,
    int kernel_length
)
{
    dim3 threads(1,256);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));
    #ifndef GB
    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    #endif
    convolutionColumnsKernel<<<blocks, threads>>>(
        #ifdef GB
        d_Input,
        #endif
        d_Dst,
        imageW,
        imageH,
        kernel_length

    );
    
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
    
#ifndef GB
    checkCudaErrors(hipUnbindTexture(texSrc));
    #endif
}


#endif
#ifdef OLD
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include "convolutionTexture_common.h"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
//Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

//Use unrolled innermost convolution loop
#define UNROLL_INNER 1

//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b)
{
    return (a % b != 0) ? (a - a % b + b) : a;
}



////////////////////////////////////////////////////////////////////////////////
// Convolution kernel and input array storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}

texture<float, 2, hipReadModeElementType> texSrc;

extern "C" void setInputArray(hipArray *a_Src)
{
}

extern "C" void detachInputArray(void)
{
}



////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template<int i> __device__ float convolutionRow(float x, float y)
{
    return
        tex2D(texSrc, x + (float)(KERNEL_RADIUS - i), y) * c_Kernel[i]
        + convolutionRow<i - 1>(x, y);
}

template<> __device__ float convolutionRow<-1>(float x, float y)
{
    return 0;
}

template<int i> __device__ float convolutionColumn(float x, float y)
{
    return
        tex2D(texSrc, x, y + (float)(KERNEL_RADIUS - i)) * c_Kernel[i]
        + convolutionColumn<i - 1>(x, y);
}

template<> __device__ float convolutionColumn<-1>(float x, float y)
{
    return 0;
}



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionRow<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x + (float)k, y) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}


extern "C" void convolutionRowsGPU(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH
)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    convolutionRowsKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");

    checkCudaErrors(hipUnbindTexture(texSrc));
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionColumn<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x, y + (float)k) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}

extern "C" void convolutionColumnsGPU(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH
)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    checkCudaErrors(hipBindTextureToArray(texSrc, a_Src));
    convolutionColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");

    checkCudaErrors(hipUnbindTexture(texSrc));
}

#endif