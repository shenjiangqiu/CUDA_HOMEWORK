#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// Utility and system includes
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples
#include<cstdlib>
#include<cstdio>
#include<iostream>
using namespace std;
// project include
#include "histogram_common.h"
#include<parseOprand.hpp>
#include<log.hpp>

#define MERGE_THREADBLOCK_SIZE 256
__global__ void naiveKernel64(unsigned int *histo,unsigned char *data,int dim){
    
    int allThreads=gridDim.x*blockDim.x;
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    
    if(index<64){
        histo[index]=0;
    }
    int numbers=(dim+allThreads-1)/allThreads;
    int base=index*numbers;
    for(int i=0;i<numbers;i++){
        int curr=base+i;
        if(curr<dim){
            unsigned char t_data=data[curr];

            unsigned int pos=(t_data>>2)&0x3FU;

            atomicAdd(histo+pos,1);
        }
    }

}

__global__ void baseKernel64(unsigned int *histo,unsigned char *d_data,int dim){
    unsigned allThreads=gridDim.x*blockDim.x;
    unsigned index=threadIdx.x+blockIdx.x*blockDim.x;
    
    if(index<64){
        histo[index]=0;
    }//reset to 0;
    int i=0;
    while(index<dim){
        unsigned char t_data=d_data[index];

        unsigned int pos=(t_data>>2)&0x3FU;

        atomicAdd(histo+pos,1);
        index+=allThreads;
    }
}
__global__ void baseKernel64_share(unsigned int *partial_histo,unsigned char *d_data,int dim){
    __shared__ unsigned share_hist[64];
    if(threadIdx.x<64){
        share_hist[threadIdx.x]=0;//init
    }
    unsigned allThreads=gridDim.x*blockDim.x;
    unsigned index=threadIdx.x+blockIdx.x*blockDim.x;
    

    int i=0;
    while(index<dim){
        unsigned char t_data=d_data[index];

        unsigned int pos=(t_data>>2)&0x3FU;

        atomicAdd(share_hist+pos,1);
        index+=allThreads;
    }
    __syncthreads(); 
    if(threadIdx.x<HISTOGRAM64_BIN_COUNT)
        partial_histo[blockIdx.x * HISTOGRAM64_BIN_COUNT + threadIdx.x] = share_hist[threadIdx.x];
    

}

__global__ void histogram64Kernel_private(uint *d_PartialHistograms, uchar *d_Data, uint dataCount)
{
    // Handle to thread block group
    
    //Per-warp subhistogram storage
    unsigned all_threads=gridDim.x*blockDim.x;
    __shared__ uint s_Hist[6*64];//every warp have a private histogram bin
    uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM64_BIN_COUNT;//the warps start point

    //Clear shared memory storage for current threadblock before processing
#pragma unroll

    int curr_index=threadIdx.x;
    while(curr_index<6*64){//totoal size of shared memory
        s_Hist[curr_index]=0;
        curr_index+=blockDim.x;
    }



    __syncthreads();

    curr_index=threadIdx.x;
    while(curr_index<dataCount){
        unsigned char data=d_Data[curr_index];
        atomicAdd(s_WarpHist+((data>>2)&0x3FU),1);
        curr_index+=all_threads;
    }

    //Merge per-warp histograms into per-block and write to global memory
    __syncthreads();
    

    for (uint bin = threadIdx.x; bin < 64; bin += 6*32)
    {
        uint sum = 0;

        for (uint i = 0; i < WARP_COUNT; i++)
        {
            sum += s_Hist[bin + i * 64] ;
        }

        d_PartialHistograms[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum;
    }
}


__global__ void mergeHistogram64Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
)
{
    //in this kernel ,each block culculate one byte in bin,we need 64 blocks,and 256 threads,each thread read seperate

    // Handle to thread block group
    //cg::thread_block cta = cg::this_thread_block();

    uint sum = 0;

    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)//this read is not coalesed
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}


void histogram64(unsigned int *d_Histogram,unsigned char *d_Data,unsigned int byteCount,unsigned* partial_histo=nullptr){
    const int blockSize=6*32;//6 warp per block
    const int gridSize=240;
    #ifdef K1
    QDEBUG("enter naive");
    naiveKernel64<<<gridSize,blockSize>>>(d_Histogram,d_Data,byteCount);
    return;
    #endif

    #ifdef K2
    QDEBUG("enter base");
    baseKernel64<<<gridSize,blockSize>>>(d_Histogram,d_Data,byteCount);
    //mergeHistogram64Kernel<<<64,MERGE_THREADBLOCK_SIZE>>>(d_Histogram,partial_histo,gridSize);
    return;
    #endif

    #ifdef K3
    QDEBUG("enter base_share");
    baseKernel64_share<<<gridSize,blockSize>>>(partial_histo,d_Data,byteCount);
    
    mergeHistogram64Kernel<<<64,MERGE_THREADBLOCK_SIZE>>>(d_Histogram,partial_histo,gridSize);
    return;
    #endif
    #ifdef K4
    QDEBUG("enter private kernel")
    histogram64Kernel_private<<<gridSize,blockSize>>>(partial_histo,d_Data,byteCount);
    
    mergeHistogram64Kernel<<<64,MERGE_THREADBLOCK_SIZE>>>(d_Histogram,partial_histo,gridSize);
    return;
    #endif

    QERROR("NO Kernel selected!");
}

int main(int argc,char**argv){
    int dim=0;
    if(0!=parseOpt(argc,(const char**)argv,dim)){
        QERROR("can not parse oprand,exit");
        return -1;
    }
    uchar *h_Data;
    uint  *h_HistogramCPU, *h_HistogramGPU;
    uchar *d_Data;
    uint  *d_Histogram,*d_partial_histo;
    StopWatchInterface *hTimer = nullptr;
    int PassFailFlag = 1;
    uint byteCount = dim;//modified,the byteCount can be arbitary number
    uint uiSizeMult = 1;

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory.\n");
    h_Data         = new uchar[byteCount];
    h_HistogramCPU = new uint[HISTOGRAM256_BIN_COUNT];
    h_HistogramGPU = new uint[HISTOGRAM256_BIN_COUNT];
    printf("...generating input data\n");
    srand(2009);

    for (uint i = 0; i < byteCount; i++)
    {
        h_Data[i] = rand() % 256;
    }
    for(uint i=0;i<64;i++){
        h_HistogramGPU[i]=0;
    }

    printf("...allocating GPU memory and copying input data\n\n");
    checkCudaErrors(hipMalloc((void **)&d_Data, byteCount));
    checkCudaErrors(hipMalloc((void **)&d_Histogram, 64 * sizeof(uint)));
    #ifdef K3
    checkCudaErrors(hipMalloc((void **)&d_partial_histo,240*64*sizeof(uint)));
    #endif
    checkCudaErrors(hipMemcpy(d_Data, h_Data, byteCount, hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(d_Histogram, h_HistogramGPU, 64*sizeof(unsigned int), hipMemcpyHostToDevice));
    
    
    
    //start the kernel

    
    histogram64(d_Histogram,d_Data,byteCount,d_partial_histo);//warm up

    hipDeviceSynchronize();
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for(int i=0;i<16;i++)//run test for 16 times
        histogram64(d_Histogram,d_Data,byteCount,d_partial_histo);
    hipDeviceSynchronize();

    sdkStopTimer(&hTimer);
    double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double)1;
    printf("histogram64() time (average) : %.5f sec, %.4f MB/sec\n\n", dAvgSecs, ((double)byteCount * 1.0e-6) / dAvgSecs);
    printf("histogram64, Throughput = %.4f MB/s, Time = %.5f s, Size = %u Bytes, NumDevsUsed = %u, Workgroup = %u\n",
           (1.0e-6 * (double)byteCount / dAvgSecs), dAvgSecs, byteCount, 1, HISTOGRAM64_THREADBLOCK_SIZE);

    printf("\nValidating GPU results...\n");
    printf(" ...reading back GPU results\n");
    checkCudaErrors(hipMemcpy(h_HistogramGPU, d_Histogram, HISTOGRAM64_BIN_COUNT * sizeof(uint), hipMemcpyDeviceToHost));

    printf(" ...histogram64CPU()\n");
    histogram64CPU(
        h_HistogramCPU,
        h_Data,
        byteCount
    );

    printf(" ...comparing the results...\n");

    for (uint i = 0; i < HISTOGRAM64_BIN_COUNT; i++)
        if (h_HistogramGPU[i] != h_HistogramCPU[i])
        {
            PassFailFlag = 0;
        }

    printf(PassFailFlag ? " ...64-bin histograms match\n\n" : " ***64-bin histograms do not match!!!***\n\n");

    printf("Shutting down 64-bin histogram...\n\n\n");


}